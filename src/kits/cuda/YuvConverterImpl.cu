#include "hip/hip_runtime.h"
#include "YuvConverterImpl.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

__constant__ float yuv2rgb[9] = {1.164f, 0.000f, 1.596f, 1.164f, -0.392f, -0.813f, 1.164f, 2.017f, 0.000f};

__constant__ float rgb2yuv[9] = {0.257f, 0.504f, 0.098f, -0.148f, -0.291f, 0.439f, 0.439f, -0.368f, -0.071f};

// YUV转BGR的CUDA核函数
__global__ void YV12ToBGR_Kernel(const uint8_t *y_data, const uint8_t *u, const uint8_t *v, uint8_t *bgr, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y_idx >= height)
        return;

    // 计算UV分量索引 (YV12格式UV分量是下采样2x2的)
    int uv_x = x / 2;
    int uv_y = y_idx / 2;
    int uv_index = uv_y * (width / 2) + uv_x;

    // 读取YUV分量并归一化
    float Y = y_data[y_idx * width + x] - 16.0f; // 使用修改后的参数名
    float U = u[uv_index] - 128.0f;
    float V = v[uv_index] - 128.0f;

    // 矩阵乘法计算RGB
    float r = Y * yuv2rgb[0] + U * yuv2rgb[1] + V * yuv2rgb[2];
    float g = Y * yuv2rgb[3] + U * yuv2rgb[4] + V * yuv2rgb[5];
    float b = Y * yuv2rgb[6] + U * yuv2rgb[7] + V * yuv2rgb[8];

    // 裁剪并转换回uint8
    int index = (y_idx * width + x) * 3; // 使用修改后的变量名
    bgr[index] = static_cast<uint8_t>(fminf(fmaxf(b, 0.0f), 255.0f));
    bgr[index + 1] = static_cast<uint8_t>(fminf(fmaxf(g, 0.0f), 255.0f));
    bgr[index + 2] = static_cast<uint8_t>(fminf(fmaxf(r, 0.0f), 255.0f));
}

// BGR转YUV的CUDA核函数
__global__ void BGRToYV12_Kernel(const uint8_t *bgr, uint8_t *y, uint8_t *u, uint8_t *v, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y_idx >= height)
        return;

    int index = (y_idx * width + x) * 3;
    float B = bgr[index];
    float G = bgr[index + 1];
    float R = bgr[index + 2];

    // 计算YUV分量
    float Y = R * rgb2yuv[0] + G * rgb2yuv[1] + B * rgb2yuv[2] + 16.0f;
    float U = R * rgb2yuv[3] + G * rgb2yuv[4] + B * rgb2yuv[5] + 128.0f;
    float V = R * rgb2yuv[6] + G * rgb2yuv[7] + B * rgb2yuv[8] + 128.0f;

    // 写入Y分量
    y[y_idx * width + x] = static_cast<uint8_t>(fminf(fmaxf(Y, 0.0f), 255.0f));

    // 只处理偶数位置像素来计算UV分量 (2x2下采样)
    if (x % 2 == 0 && y_idx % 2 == 0)
    {
        int uv_index = (y_idx / 2) * (width / 2) + (x / 2);
        u[uv_index] = static_cast<uint8_t>(fminf(fmaxf(U, 0.0f), 255.0f));
        v[uv_index] = static_cast<uint8_t>(fminf(fmaxf(V, 0.0f), 255.0f));
    }
}

// YuvConverterImpl::YuvConverterImpl(int width, int height)
//     : width_(width), height_(height)
// {
//     // 分配设备内存
//     hipMalloc(&d_yv12_, width * height * 3 / 2);  // YV12大小: w*h + w*h/4 + w*h/4
//     hipMalloc(&d_bgr_, width * height * 3);       // BGR大小: w*h*3
//     hipMalloc(&d_temp_, width * height * sizeof(float) * 3); // 临时内存
// }
// YuvConverterImpl::~YuvConverterImpl() {
//     hipFree(d_yv12_);
//     hipFree(d_bgr_);
//     hipFree(d_temp_);
// }
int initCudaMemory(uint8_t **d_yv12, uint8_t **d_bgr, uint8_t **d_rgb, float **d_temp, int w, int h)
{
    hipMalloc((void **)d_yv12, w * h * 3 / 2);
    hipMalloc((void **)d_bgr, w * h * 3);
    hipMalloc((void **)d_rgb, w * h * 3);
    hipMalloc((void **)d_temp, w * h * sizeof(float) * 3);
    return hipGetLastError(); // 返回错误码
}

void freeCudaMemory(uint8_t *&d_yv12, uint8_t *&d_bgr, uint8_t *&d_rgb, float *&d_temp)
{
    if (d_yv12)
    {
        hipFree(d_yv12);
        d_yv12 = nullptr;
    }
    if (d_bgr)
    {
        hipFree(d_bgr);
        d_bgr = nullptr;
    }
    if (d_rgb)
    {
        hipFree(d_rgb);
        d_rgb = nullptr;
    }
    if (d_temp)
    {
        hipFree(d_temp);
        d_temp = nullptr;
    }
}

int YV12ToBGR(const uint8_t *yv12Data, uint8_t *bgrData, uint8_t *d_yv12_, uint8_t *d_bgr_, int width_, int height_)
{
    // 将YV12数据拷贝到设备
    hipMemcpy(d_yv12_, yv12Data, width_ * height_ * 3 / 2, hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    dim3 block(16, 16);
    dim3 grid((width_ + block.x - 1) / block.x, (height_ + block.y - 1) / block.y);

    // 获取YUV分量指针
    uint8_t *d_y = d_yv12_;
    uint8_t *d_u = d_y + width_ * height_;
    uint8_t *d_v = d_u + (width_ * height_) / 4;

    // 调用核函数
    YV12ToBGR_Kernel<<<grid, block>>>(d_y, d_u, d_v, d_bgr_, width_, height_);

    // 将结果拷贝回主机
    hipMemcpy(bgrData, d_bgr_, width_ * height_ * 3, hipMemcpyDeviceToHost);
    return 0; // 返回成功
}

// BGR转YV12实现
int BGRToYV12(const uint8_t *bgrData, uint8_t *yv12Data, uint8_t *d_bgr_, uint8_t *d_yv12_, int width_, int height_)
{
    // 将BGR数据拷贝到设备
    hipMemcpy(d_bgr_, bgrData, width_ * height_ * 3, hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    dim3 block(16, 16);
    dim3 grid((width_ + block.x - 1) / block.x, (height_ + block.y - 1) / block.y);

    // 获取YUV分量指针
    uint8_t *d_y = d_yv12_;
    uint8_t *d_u = d_y + width_ * height_;
    uint8_t *d_v = d_u + (width_ * height_) / 4;

    // 调用核函数
    BGRToYV12_Kernel<<<grid, block>>>(d_bgr_, d_y, d_u, d_v, width_, height_);

    // 将结果拷贝回主机
    hipMemcpy(yv12Data, d_yv12_, width_ * height_ * 3 / 2, hipMemcpyDeviceToHost);
    return 0;
}

// ============== YV12 -> RGB =================
__global__ void YV12ToRGB_Kernel(const uint8_t *y_data,
                                 const uint8_t *u,
                                 const uint8_t *v,
                                 uint8_t *rgb, // 输出：R→G→B
                                 int width,
                                 int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int uv_x = x >> 1;
    int uv_y = y >> 1;
    int uv_idx = uv_y * (width >> 1) + uv_x;

    float Y = y_data[y * width + x] - 16.0f;
    float U = u[uv_idx] - 128.0f;
    float V = v[uv_idx] - 128.0f;

    float r = Y * yuv2rgb[0] + U * yuv2rgb[1] + V * yuv2rgb[2];
    float g = Y * yuv2rgb[3] + U * yuv2rgb[4] + V * yuv2rgb[5];
    float b = Y * yuv2rgb[6] + U * yuv2rgb[7] + V * yuv2rgb[8];

    int idx = (y * width + x) * 3;
    rgb[idx] = static_cast<uint8_t>(fminf(fmaxf(r, 0.0f), 255.0f));     // R
    rgb[idx + 1] = static_cast<uint8_t>(fminf(fmaxf(g, 0.0f), 255.0f)); // G
    rgb[idx + 2] = static_cast<uint8_t>(fminf(fmaxf(b, 0.0f), 255.0f)); // B
}

int YV12ToRGB(const uint8_t *yv12Data, uint8_t *rgbData, uint8_t *d_yv12_, uint8_t *d_rgb_, int width, int height)
{
    hipMemcpy(d_yv12_, yv12Data, width * height * 3 / 2, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    uint8_t *d_y = d_yv12_;
    uint8_t *d_u = d_y + width * height;
    uint8_t *d_v = d_u + (width * height) / 4;

    YV12ToRGB_Kernel<<<grid, block>>>(d_y, d_u, d_v, d_rgb_, width, height);

    hipMemcpy(rgbData, d_rgb_, width * height * 3, hipMemcpyDeviceToHost);
    return 0;
}

// ============== RGB -> YV12 =================
__global__ void RGBToYV12_Kernel(const uint8_t *rgb, // 输入：R→G→B
                                 uint8_t *y,
                                 uint8_t *u,
                                 uint8_t *v,
                                 int width,
                                 int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y_idx >= height)
        return;

    int idx = (y_idx * width + x) * 3;
    float R = rgb[idx];
    float G = rgb[idx + 1];
    float B = rgb[idx + 2];

    float Yf = R * rgb2yuv[0] + G * rgb2yuv[1] + B * rgb2yuv[2] + 16.0f;
    float Uf = R * rgb2yuv[3] + G * rgb2yuv[4] + B * rgb2yuv[5] + 128.0f;
    float Vf = R * rgb2yuv[6] + G * rgb2yuv[7] + B * rgb2yuv[8] + 128.0f;

    y[y_idx * width + x] = static_cast<uint8_t>(fminf(fmaxf(Yf, 0.0f), 255.0f));

    if ((x & 1) == 0 && (y_idx & 1) == 0)
    {
        int uv_idx = (y_idx >> 1) * (width >> 1) + (x >> 1);
        u[uv_idx] = static_cast<uint8_t>(fminf(fmaxf(Uf, 0.0f), 255.0f));
        v[uv_idx] = static_cast<uint8_t>(fminf(fmaxf(Vf, 0.0f), 255.0f));
    }
}

int RGBToYV12(const uint8_t *rgbData, uint8_t *yv12Data, uint8_t *d_rgb_, uint8_t *d_yv12_, int width, int height)
{
    hipMemcpy(d_rgb_, rgbData, width * height * 3, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    uint8_t *d_y = d_yv12_;
    uint8_t *d_u = d_y + width * height;
    uint8_t *d_v = d_u + (width * height) / 4;

    RGBToYV12_Kernel<<<grid, block>>>(d_rgb_, d_y, d_u, d_v, width, height);

    hipMemcpy(yv12Data, d_yv12_, width * height * 3 / 2, hipMemcpyDeviceToHost);
    return 0;
}